﻿
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define MATRIX_WIDTH 1000 //dimensions of matrices
#define MATRIX_SIZE (MATRIX_WIDTH * MATRIX_WIDTH) //total number of elements in matrices
#define NBYTES (MATRIX_SIZE * sizeof(float))

//int BLOCK_WIDTH = 1;
int BLOCK_WIDTH[] = { 2, 5, 10, 25, 32 };

float M[MATRIX_SIZE];
float N[MATRIX_SIZE];
float P[MATRIX_SIZE];

//functions to be tested
void cudaTransferTest();
void cudaMatMult(float* M, float* N, float* P, int WIDTH);

//matrix multiplication kernel, called by cudaMatMult function
__global__ void matMultKernel(float* M, float* N, float* P, int WIDTH)
{
	// calculate row, col index
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < WIDTH && col < WIDTH)
	{
		float Pvalue = 0;
		//each thread computes one element of the block sub-matrix
		for (int k = 0; k < WIDTH; k++) {
			Pvalue += M[row * WIDTH + k] * N[k * WIDTH + col];
		}
		P[row * WIDTH + col] = Pvalue;
	}
}

void checkGPUanswer(float* M, float* N, float* GPU_P, int WIDTH)
{
	bool passed;
	float check;

	for (int i = 0; i < WIDTH; i++)
	{
		for (int j = 0; j < WIDTH; j++)
		{
			check = 0;

			for (int k = 0; k < WIDTH; k++)
			{
				check += M[i * WIDTH + k] * N[k * WIDTH + j];
				if (GPU_P[i * WIDTH + j] != check) passed = 0;
			}
		}
	}
	passed = 1;

	if (passed)	printf("TEST PASSED\n");
	else		printf("TEST FAILED\n");
}

//standard matrix multiplication, computed using CPU
void CPUmatMult(float* M, float* N, float* P, int WIDTH)
{
	hipEvent_t start, stop;
	float gpu_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t err;

	for (int l = 0; l < 5; l++)
	{
		hipEventRecord(start, 0); // start timer
		hipDeviceSynchronize();

		for (int i = 0; i < WIDTH; i++)
		{
			for (int j = 0; j < WIDTH; j++)
			{
				for (int k = 0; k < WIDTH; k++)
				{
					P[i * WIDTH + j] += M[i * WIDTH + k] * N[k * WIDTH + j];
				}
			}
		}

		hipEventRecord(stop, 0);	// end timer
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("Time for CPU matrix multiplication: %f\n", gpu_time); //display results
	}


}

int main()
{
	srand(time(NULL));

	hipHostMalloc((void**)&M, NBYTES, hipHostMallocDefault);
	hipHostMalloc((void**)&N, NBYTES, hipHostMallocDefault);
	hipHostMalloc((void**)&P, NBYTES, hipHostMallocDefault);

	for (int i = 0; i < MATRIX_SIZE; i++)
	{	// value between 0 and 10, one decimal place
		M[i] = rand() % 100 / (float) 10.0;
		N[i] = rand() % 100 / (float) 10.0;
		P[i] = 0.0;
	}

	//cudaTransferTest();

	//for(int i = 0; i < 5; i++)
	//{
		cudaMatMult(M, N, P, MATRIX_WIDTH);
	//	CPUmatMult(M, N, P, MATRIX_WIDTH);
	//}

	hipHostFree(M);
	hipHostFree(N);
	hipHostFree(P);

	return 0;
}

void cudaTransferTest()
{
	hipEvent_t start, stop;
	float gpu_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t err;

	float* dM, * dN, * dP;

	//allocate memory for matrices on device
	hipMalloc((void**)(&dM), NBYTES);
	hipMalloc((void**)(&dN), NBYTES);
	hipMalloc((void**)(&dP), NBYTES);

	//check memory allocation was successful
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error allocating memory in device");

	for(int i = 0; i < 5; i++)
	{
		hipEventRecord(start, 0); // start timer
		hipDeviceSynchronize();

		//copy information from host to device
		hipMemcpy(dM, M, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dN, N, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

		hipEventRecord(stop, 0);	// end timer
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("Time to send matrices to device from host: %f\n", gpu_time); //display results

		hipEventRecord(start, 0); // start timer
		hipDeviceSynchronize();

		//copy information from device to host
		hipMemcpy(M, dM, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(N, dN, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);

		hipEventRecord(stop, 0);	// end timer
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("Time to send matrices to host from device: %f\n", gpu_time); //display results


	}

	hipFree(dM);
	hipFree(dN);
	hipFree(dP);

}

void cudaMatMult(float* M, float* N, float* P, int WIDTH)
{
	hipEvent_t start, stop;
	float gpu_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t err;

	float* dM, * dN, * dP;

	//allocate memory for matrices on device
	hipMalloc((void**)(&dM), NBYTES);
	hipMalloc((void**)(&dN), NBYTES);
	hipMalloc((void**)(&dP), NBYTES);

	//check memory allocation was successful
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error allocating memory in device");

	hipMemcpy(dM, M, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dN, N, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

	for(int i = 0; i < 5; i++)
	{

		int NUM_BLOCKS = WIDTH / BLOCK_WIDTH[i];
		if (WIDTH % BLOCK_WIDTH[i]) NUM_BLOCKS++;

		dim3 dimGrid(NUM_BLOCKS, NUM_BLOCKS);
		dim3 dimBlock(BLOCK_WIDTH[i], BLOCK_WIDTH[i]);

		for (int i = 0; i < 5; i++)
		{
			hipEventRecord(start, 0); // start timer
			hipDeviceSynchronize();

			//copy information from device to host
			matMultKernel <<<dimGrid, dimBlock >>> (dM, dN, dP, WIDTH);

			hipEventRecord(stop, 0);	// end timer
			hipEventSynchronize(stop);
			hipEventElapsedTime(&gpu_time, start, stop);
			printf("Time for GPU matrix multiplication: %f\n", gpu_time); //display results
			checkGPUanswer(M, N, P, MATRIX_WIDTH);

		}
	}

	hipFree(dM);
	hipFree(dN);
	hipFree(dP);

}
