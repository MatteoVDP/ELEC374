#include "hip/hip_runtime.h"
﻿
//written by Matteo Van der Plaat (20287556)

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define MATRIX_WIDTH 100 //dimensions of matrices
#define MATRIX_SIZE (MATRIX_WIDTH * MATRIX_WIDTH) //total number of elements in matrices
#define NBYTES (MATRIX_SIZE * sizeof(float))

//int BLOCK_WIDTH = 1;
int BLOCK_WIDTH[] = { 2, 5, 10, 25, 32 };
int TILE_SIZE[] = { 2, 5, 10, 25 };
#define TILE_WIDTH 25

float M[MATRIX_SIZE];
float N[MATRIX_SIZE];
float P[MATRIX_SIZE];

//functions to be tested
void cudaTransferTest();
void cudaMatMult(float* M, float* N, float* P, int WIDTH);

//matrix multiplication kernel, called by cudaMatMult function
__global__ void tiledMatMultKernel(float* M, float* N, float* P, int WIDTH)
{
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	// calculate row, col index
	int row = by * TILE_WIDTH + ty;
	int col = bx * TILE_WIDTH + tx;

	float Pvalue = 0;

	// Loop over the M and N tiles required to compute the P element
	for (int i = 0; i < WIDTH / TILE_WIDTH; ++i)
	{
		// Collaborative loading of M and N tiles into shared memory
		Mds[ty][tx] = M[row * WIDTH + i * TILE_WIDTH + tx];
		Nds[ty][tx] = N[(i * TILE_WIDTH + ty) * WIDTH + col];

		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; ++k) // Perform one phase of dot product
			Pvalue += Mds[ty][k] * Nds[k][tx];

		__syncthreads();

	}
	P[row * WIDTH + col] = Pvalue; // All threads write to their P element

}

//function used to check validity of value outputted by GPU Mat Mult function
void checkGPUanswer(float* M, float* N, float* GPU_P, int WIDTH)
{
	bool passed;
	float check;

	//calculate correct values in CPU and compare against GPU value
	for (int i = 0; i < WIDTH; i++)
	{
		for (int j = 0; j < WIDTH; j++)
		{
			check = 0;

			for (int k = 0; k < WIDTH; k++)
			{
				check += M[i * WIDTH + k] * N[k * WIDTH + j];
				if (GPU_P[i * WIDTH + j] != check) passed = 0;
			}
		}
	}
	passed = 1; //if all values match up, test passed

	if (passed)	printf("TEST PASSED\n");
	else		printf("TEST FAILED\n");
}

//standard matrix multiplication, computed using CPU
void CPUmatMult(float* M, float* N, float* P, int WIDTH)
{
	//initialization values used for timing
	hipEvent_t start, stop;
	float gpu_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t err;

	for (int l = 0; l < 5; l++)
	{
		hipEventRecord(start, 0); // start timer
		hipDeviceSynchronize();

		for (int i = 0; i < WIDTH; i++)
		{
			for (int j = 0; j < WIDTH; j++)
			{
				for (int k = 0; k < WIDTH; k++)
				{
					P[i * WIDTH + j] += M[i * WIDTH + k] * N[k * WIDTH + j];
				}
			}
		}

		hipEventRecord(stop, 0);	// end timer
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("Time for CPU matrix multiplication: %f\n", gpu_time); //display results
	}


}

int main()
{
	srand(time(NULL)); //seed random function

	//allocate memory in host
	hipHostMalloc((void**)&M, NBYTES);
	hipHostMalloc((void**)&N, NBYTES);
	hipHostMalloc((void**)&P, NBYTES);

	for (int i = 0; i < MATRIX_SIZE; i++)
	{	// fill matrices M and N with randon values for testing
		M[i] = rand() % 100 / (float)10.0;
		N[i] = rand() % 100 / (float)10.0;
		P[i] = 0.0;
	}

	//function used for testing transferring data between host and device
	//cudaTransferTest();

	//functions used for testing matrix multiplication using GPUs and comparing against CPUs
	cudaMatMult(M, N, P, MATRIX_WIDTH); // GPU/Cuda matrix multiplication
	//CPUmatMult(M, N, P, MATRIX_WIDTH); // CPU matrix multiplication

	//free host memory 
	hipHostFree(M);
	hipHostFree(N);
	hipHostFree(P);

	return 0;
}

void cudaTransferTest()
{
	hipEvent_t start, stop;
	float gpu_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t err;

	float* dM, * dN, * dP;

	//allocate memory for matrices on device
	hipMalloc((void**)(&dM), NBYTES);
	hipMalloc((void**)(&dN), NBYTES);
	hipMalloc((void**)(&dP), NBYTES);

	//check memory allocation was successful
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error allocating memory in device");

	//repeat 5 times to ensure correctness
	for (int i = 0; i < 5; i++)
	{
		hipEventRecord(start, 0); // start timer
		hipDeviceSynchronize();

		//copy information from host to device
		hipMemcpy(dM, M, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dN, N, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

		hipEventRecord(stop, 0); // end timer
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("Time to send matrices to device from host: %f\n", gpu_time); //display results

		hipEventRecord(start, 0); // start timer
		hipDeviceSynchronize();

		//copy information from device to host
		hipMemcpy(M, dM, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(N, dN, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);

		hipEventRecord(stop, 0);	// end timer
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("Time to send matrices to host from device: %f\n", gpu_time); //display results


	}
	//free device memory
	hipFree(dM);
	hipFree(dN);
	hipFree(dP);

}

void cudaMatMult(float* M, float* N, float* P, int WIDTH)
{
	hipEvent_t start, stop;
	float gpu_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t err;

	float* dM, * dN, * dP;

	//allocate memory for matrices on device
	hipMalloc((void**)(&dM), NBYTES);
	hipMalloc((void**)(&dN), NBYTES);
	hipMalloc((void**)(&dP), NBYTES);

	//check memory allocation was successful
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error allocating memory in device");

	//int NUM_TILES = WIDTH / TILE_WIDTH[i];
	//if (WIDTH % TILE_WIDTH[i]) NUM_TILES++;

	//define dimensions of grid and blocks
	dim3 dimGrid(WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
		 
	//dim3 dimGrid(NUM_BLOCKS, NUM_BLOCKS);
	//dim3 dimBlock(BLOCK_WIDTH[i], BLOCK_WIDTH[i]);



	for (int j = 0; j < 5; j++)
	{

		//copy memory from host to device
		hipMemcpy(dM, M, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dN, N, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

		hipEventRecord(start, 0); // start timer
		hipDeviceSynchronize();

		//calculate matrix multiplication using Cuda and GPUs,, enabling synchronization
		tiledMatMultKernel << <dimGrid, dimBlock >> > (dM, dN, dP, WIDTH);

		hipEventRecord(stop, 0); // end timer
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("Time for GPU matrix multiplication: %f\n", gpu_time); //display results

		hipMemcpy(M, dM, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(N, dN, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);

		//checkGPUanswer(M, N, P, MATRIX_WIDTH); //make sure answers are correct by comparing against CPU values

	}

	//free device memory
	hipFree(dM);
	hipFree(dN);
	hipFree(dP);

}
